#include "hip/hip_runtime.h"
/*
 *  This file is part of NESTGPU.
 *
 *  Copyright (C) 2021 The NEST Initiative
 *
 *  NESTGPU is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  NESTGPU is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with NESTGPU.  If not, see <http://www.gnu.org/licenses/>.
 *
 */





#include <config.h>
#include <stdio.h>
#include "send_spike.h"
#include "cuda_error.h"

int *d_SpikeNum;
int *d_SpikeSourceIdx;
int *d_SpikeConnIdx;
float *d_SpikeHeight;
int *d_SpikeTargetNum;

int *d_PoissonSpikeNum;
int *d_PoissonSpikeSourceIdx;
int *d_PoissonSpikeConnIdx;
float *d_PoissonSpikeHeight;
int *d_PoissonSpikeTargetNum;

__device__ int MaxSpikeNum;
__device__ int *SpikeNum;
__device__ int *SpikeSourceIdx;
__device__ int *SpikeConnIdx;
__device__ float *SpikeHeight;
__device__ int *SpikeTargetNum;

__device__ int *PoissonSpikeNum;
__device__ int *PoissonSpikeSourceIdx;
__device__ int *PoissonSpikeConnIdx;
__device__ float *PoissonSpikeHeight;
__device__ int *PoissonSpikeTargetNum;

__device__ void SendSpike(int i_source, int i_conn, float height,
			  int target_num)
{
  int pos = atomicAdd(SpikeNum, 1);
  if (pos>=MaxSpikeNum) {
    printf("Number of spikes larger than MaxSpikeNum: %d\n", MaxSpikeNum);
    *SpikeNum = MaxSpikeNum;
    return;
  }
  SpikeSourceIdx[pos] = i_source;
  SpikeConnIdx[pos] = i_conn;
  SpikeHeight[pos] = height;
  SpikeTargetNum[pos] = target_num;
}

__device__ void PoissonSendSpike(int i_source, int i_conn, float height,
				 int target_num)
{
  int pos = atomicAdd(PoissonSpikeNum, 1);
  if (pos>=MaxSpikeNum) {
    printf("Number of spikes larger than MaxSpikeNum: %d\n", MaxSpikeNum);
    *PoissonSpikeNum = MaxSpikeNum;
    return;
  }
  PoissonSpikeSourceIdx[pos] = i_source;
  PoissonSpikeConnIdx[pos] = i_conn;
  PoissonSpikeHeight[pos] = height;
  PoissonSpikeTargetNum[pos] = target_num;
}

__global__ void DeviceSpikeInit(int *spike_num, int *spike_source_idx,
				int *spike_conn_idx, float *spike_height,
				int *spike_target_num, int *poiss_spike_num,
				int *poiss_spike_source_idx,
				int *poiss_spike_conn_idx,
				float *poiss_spike_height,
				int *poiss_spike_target_num,
				int max_spike_num)
{
  SpikeNum = spike_num;
  SpikeSourceIdx = spike_source_idx;
  SpikeConnIdx = spike_conn_idx;
  SpikeHeight = spike_height;
  SpikeTargetNum = spike_target_num;

  PoissonSpikeNum = poiss_spike_num;
  PoissonSpikeSourceIdx = poiss_spike_source_idx;
  PoissonSpikeConnIdx = poiss_spike_conn_idx;
  PoissonSpikeHeight = poiss_spike_height;
  PoissonSpikeTargetNum = poiss_spike_target_num;

  MaxSpikeNum = max_spike_num;

  *SpikeNum = 0;
  *PoissonSpikeNum = 0;
}


void SpikeInit(int max_spike_num)
{
  //h_SpikeTargetNum = new int[PrefixScan::AllocSize];

  gpuErrchk(hipMalloc(&d_SpikeNum, sizeof(int)));
  gpuErrchk(hipMalloc(&d_SpikeSourceIdx, max_spike_num*sizeof(int)));
  gpuErrchk(hipMalloc(&d_SpikeConnIdx, max_spike_num*sizeof(int)));
  gpuErrchk(hipMalloc(&d_SpikeHeight, max_spike_num*sizeof(float)));
  gpuErrchk(hipMalloc(&d_SpikeTargetNum, max_spike_num*sizeof(int)));
  
  gpuErrchk(hipMalloc(&d_PoissonSpikeNum, sizeof(int)));
  gpuErrchk(hipMalloc(&d_PoissonSpikeSourceIdx, max_spike_num*sizeof(int)));
  gpuErrchk(hipMalloc(&d_PoissonSpikeConnIdx, max_spike_num*sizeof(int)));
  gpuErrchk(hipMalloc(&d_PoissonSpikeHeight, max_spike_num*sizeof(float)));
  gpuErrchk(hipMalloc(&d_PoissonSpikeTargetNum, max_spike_num*sizeof(int)));
  
  //printf("here: SpikeTargetNum size: %d", max_spike_num);
  DeviceSpikeInit<<<1,1>>>(d_SpikeNum, d_SpikeSourceIdx, d_SpikeConnIdx,
			   d_SpikeHeight, d_SpikeTargetNum, d_PoissonSpikeNum,
			   d_PoissonSpikeSourceIdx, d_PoissonSpikeConnIdx,
			   d_PoissonSpikeHeight, d_PoissonSpikeTargetNum,
			   max_spike_num);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
}

__global__ void SpikeReset()
{
  *SpikeNum = 0;
  *PoissonSpikeNum = 0;
}
